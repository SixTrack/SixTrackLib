#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/cuda/wrappers/controller_wrappers.h"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

#if !defined( SIXTRL_NO_SYSTEM_INCLUDES )
    #include <hip/hip_runtime_api.h>
#endif /* !defined( SIXTRL_NO_SYSTEM_INCLUDES ) */

#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/common/definitions.h"
    #include "sixtracklib/common/buffer/buffer_type.h"
    #include "sixtracklib/common/control/definitions.h"
//     #include "sixtracklib/common/control/kernel_config_base.h"

    #include "sixtracklib/cuda/definitions.h"
    #include "sixtracklib/cuda/control/kernel_config.h"
    #include "sixtracklib/cuda/kernels/managed_buffer_remap.cuh"
    #include "sixtracklib/cuda/argument.h"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

void NS(Buffer_remap_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT buffer_arg )
{
    dim3 const* ptr_blocks = SIXTRL_NULLPTR;
        NS(CudaKernelConfig_get_ptr_const_blocks)( kernel_config );

    dim3 const* ptr_threads = SIXTRL_NULLPTR;
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    /* kernel config */

    SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

    SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
        NS(ARCHITECTURE_CUDA) );

    SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );

    /* argument: */

    SIXTRL_ASSERT( buffer_arg != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( NS(Argument_get_arch_id)( buffer_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

    SIXTRL_ASSERT( NS(Argument_has_argument_buffer)( buffer_arg) );
    SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( buffer_arg ) );
    SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
        buffer_arg ) != SIXTRL_NULLPTR );

    SIXTRL_ASSERT( NS(Argument_get_const_ptr_base_controller)(
        buffer_arg ) != SIXTRL_NULLPTR );

    SIXTRL_ASSERT( NS(Argument_get_cobjects_buffer_slot_size)( buffer_arg ) >
                  ( NS(buffer_size_t) )0u );

    NS(ManagedBuffer_remap_cuda)<<< *ptr_blocks, *ptr_threads >>>(
        NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)( buffer_arg ),
        NS(Argument_get_cobjects_buffer_slot_size)( buffer_arg ) );
}

void NS(Buffer_remap_cuda_debug_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT conf,
    NS(CudaArgument)* SIXTRL_RESTRICT buffer_arg,
    NS(cuda_arg_buffer_t) SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = SIXTRL_NULLPTR;
    dim3 const* ptr_threads = SIXTRL_NULLPTR;

    if( ( NS(KernelConfig_get_arch_id)( conf ) == NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( conf ) ) )
    {
        ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)( conf );

        ptr_threads =
            NS(CudaKernelConfig_get_ptr_const_threads_per_block)( conf );
    }

    if( ( ptr_blocks  != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( buffer_arg != SIXTRL_NULLPTR ) &&
        ( NS(Argument_has_argument_buffer)( buffer_arg ) ) &&
        ( NS(Argument_uses_cobjects_buffer)( buffer_arg ) ) &&
        ( NS(Argument_get_const_cobjects_buffer)( buffer_arg ) !=
          SIXTRL_NULLPTR ) &&
        ( NS(Argument_get_const_ptr_base_controller)( buffer_arg) !=
          SIXTRL_NULLPTR ) &&
        ( NS(Argument_get_cobjects_buffer_slot_size)( buffer_arg ) >
          ( NS(arch_size_t) )0u ) &&
        ( dbg_register_arg != SIXTRL_NULLPTR ) &&
        ( NS(Argument_has_argument_buffer)( dbg_register_arg ) ) &&
        ( NS(Argument_uses_raw_argument)( dbg_register_arg ) ) &&
        ( NS(Argument_get_ptr_raw_argument)( dbg_register_arg ) ) &&
        ( NS(Argument_get_size)( dbg_register_arg ) ==
          sizeof( NS(arch_debugging_t) ) ) &&
        ( NS(Argument_get_const_ptr_base_controller)( dbg_register_arg ) ==
          NS(Argument_get_const_ptr_base_controller)( buffer_arg ) ) )
    {
        NS(ManagedBuffer_remap_cuda_debug)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                buffer_arg ),
        NS(Argument_get_cobjects_buffer_slot_size)( buffer_arg ),
        reinterpret_cast< NS(arch_debugging_t)* >( dbg_register_arg ) );
    }
}

/* end: sixtracklib/cuda/wrappers/controller_wrappers.cu */
