#include "sixtracklib/cuda/wrappers/argument_operations.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "sixtracklib/common/definitions.h"
#include "sixtracklib/common/context/definitions.h"
#include "sixtracklib/cuda/definitions.h"

NS(cuda_arg_buffer_t) NS(CudaArgument_alloc_arg_buffer)(
    NS(context_size_t) const capacity )
{
    NS(cuda_arg_buffer_t) arg_buffer = SIXTRL_NULLPTR;

    if( capacity > ( NS(context_size_t) )0u )
    {
        hipError_t const ret = hipMalloc( ( void** )&arg_buffer, capacity );

        if( ret != hipSuccess )
        {
            if( arg_buffer != SIXTRL_NULLPTR )
            {
                hipFree( arg_buffer );
                arg_buffer = SIXTRL_NULLPTR;
            }
        }
    }

    return arg_buffer;
}

void NS(CudaArgument_free_arg_buffer)(
    NS(cuda_arg_buffer_t) SIXTRL_RESTRICT arg_buffer )
{
    if( arg_buffer != SIXTRL_NULLPTR )
    {
        hipFree( arg_buffer );
        arg_buffer = SIXTRL_NULLPTR;
    }

    return;
}

/* end: sixtracklib/cuda/wrappers/argument_operations.cu */
