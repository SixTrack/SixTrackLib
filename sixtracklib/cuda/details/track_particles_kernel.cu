#include "hip/hip_runtime.h"
#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/cuda/impl/track_particles_kernel.cuh"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

#if !defined( SIXTRL_NO_SYSTEM_INCLUDES )
    #include <stddef.h>
    #include <stdint.h>
    #include <stdlib.h>

    #include <hip/hip_runtime.h>
    #include <hip/hip_runtime.h>
#endif /* !defined( SIXTRL_NO_SYSTEM_INCLUDES ) */

#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/_impl/definitions.h"
    #include "sixtracklib/common/impl/buffer_defines.h"
    #include "sixtracklib/common/impl/managed_buffer_minimal.h"
    #include "sixtracklib/common/impl/managed_buffer_remap.h"
    #include "sixtracklib/common/impl/particles_defines.h"
    #include "sixtracklib/common/particles.h"
    #include "sixtracklib/common/track.h"
    #include "sixtracklib/cuda/impl/cuda_tools.h"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

__global__ void NS(Remap_particles_beam_elements_buffers_kernel_cuda)(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT particles_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT beam_elem_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag )
{
    typedef NS(buffer_size_t) buf_size_t;

    size_t const thread_id = NS(Cuda_get_1d_thread_id_in_kernel)();
    size_t const total_num_threads = NS(Cuda_get_total_num_threads_in_kernel)();

    size_t const thread_id_to_remap_particles_buffers = ( size_t )0u;

    size_t const thread_id_to_remap_beam_elements_buffers =
        ( total_num_threads > ( size_t )1u )
            ? ( thread_id_to_remap_particles_buffers + ( size_t )1u )
            : ( thread_id_to_remap_particles_buffers );

    if( thread_id <= thread_id_to_remap_beam_elements_buffers )
    {
        buf_size_t const slot_size = ( buf_size_t )8u;
        int success_flag = ( int )0u;

        if( thread_id == thread_id_to_remap_particles_buffers )
        {
            if( ( particles_buffer != SIXTRL_NULLPTR ) &&
                ( particles_buffer != beam_elem_buffer ) )
            {
                if( 0 != NS(ManagedBuffer_remap)( particles_buffer, slot_size ) )
                {
                    success_flag |= -2;
                }
            }
            else
            {
                success_flag |= -1;
            }
        }

        if( thread_id == thread_id_to_remap_beam_elements_buffers )
        {
            if( ( beam_elem_buffer != SIXTRL_NULLPTR ) &&
                ( beam_elem_buffer != particles_buffer ) )
            {
                if( 0 != NS(ManagedBuffer_remap)(
                        beam_elem_buffer, slot_size ) )
                {
                    success_flag |= -4;
                }
            }
            else
            {
                success_flag |= -1;
            }
        }

        if( ( success_flag != 0 ) && ( ptr_success_flag != SIXTRL_NULLPTR ) )
        {
            #if ( defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) )
            atomicOr( ptr_success_flag, success_flag );
            #else  /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) */
            *ptr_success_flag |= success_flag;
            #endif /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) */
        }
    }

    return;
}


__global__ void NS(Track_particles_beam_elements_kernel_cuda)(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT particles_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char const* SIXTRL_RESTRICT beam_elem_buffer,
    SIXTRL_UINT64_T const num_turns,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag )
{
    typedef NS(buffer_size_t) buf_size_t;
    typedef SIXTRL_BUFFER_OBJ_ARGPTR_DEC  NS(Object)*        obj_iter_t;
    typedef SIXTRL_BUFFER_OBJ_ARGPTR_DEC  NS(Object) const*  obj_const_iter_t;
    typedef SIXTRL_BUFFER_DATAPTR_DEC     NS(Particles)*     ptr_particles_t;

    int32_t success_flag = ( int32_t )0u;
    buf_size_t const slot_size = ( buf_size_t )8u;

    if( ( !NS(ManagedBuffer_needs_remapping( particles_buffer, slot_size ) ) ) &&
        ( !NS(ManagedBuffer_needs_remapping( beam_elem_buffer, slot_size ) ) ) )
    {
        size_t global_particle_id = NS(Cuda_get_1d_thread_id_in_kernel)();
        size_t const stride = NS(Cuda_get_1d_thread_stride_in_kernel)();

        size_t object_begin_particle_id = ( size_t )0u;

        obj_iter_t part_block_it  = NS(ManagedBuffer_get_objects_index_begin)(
                particles_buffer, slot_size );

        obj_iter_t part_block_end = NS(ManagedBuffer_get_objects_index_end)(
                particles_buffer, slot_size );

        obj_const_iter_t be_begin = NS(ManagedBuffer_get_const_objects_index_begin)(
                beam_elem_buffer, slot_size );

        obj_const_iter_t be_end = NS(ManagedBuffer_get_const_objects_index_end)(
                beam_elem_buffer, slot_size );

        for( ; part_block_it != part_block_end ; ++part_block_it )
        {
            ptr_particles_t particles = ( ptr_particles_t )(
                uintptr_t )NS(Object_get_begin_addr)( part_block_it );

            size_t const object_end_particle_id = object_begin_particle_id +
                NS(Particles_get_num_of_particles)( particles );

            SIXTRL_ASSERT( NS(Object_get_type_id)( part_block_it ) ==
                           NS(OBJECT_TYPE_PARTICLE) );

            if( ( global_particle_id <  object_end_particle_id   ) &&
                ( global_particle_id >= object_begin_particle_id ) )
            {
                size_t const particle_id =
                    global_particle_id - object_begin_particle_id;

                SIXTRL_UINT64_T turn = ( SIXTRL_UINT64_T )0u;

                SIXTRL_ASSERT( particle_id <
                    NS(Particles_get_num_of_particles)( particles ) );

                for( ; turn < num_turns ; ++turn )
                {
                    success_flag |= NS(Track_particle_beam_elements)(
                        particles, particle_id, be_begin, be_end );
                }
            }

            object_begin_particle_id = object_end_particle_id;
        }
    }
    else
    {
        if( NS(ManagedBuffer_needs_remapping( particles_buffer, slot_size ) ) )
        {
            success_flag |= -2;
        }

        if( NS(ManagedBuffer_needs_remapping( beam_elem_buffer, slot_size ) ) )
        {
            success_flag |= -4;
        }
    }

    if( ( success_flag != 0 ) && ( ptr_success_flag != SIXTRL_NULLPTR ) )
    {
        #if ( defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) )
        atomicOr( ptr_success_flag, success_flag );
        #else  /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) */
        *ptr_success_flag |= success_flag;
        #endif /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ >= 120 ) */
    }

    return;
}

/* end: sixtracklib/cuda/details/track_particles_kernel.cu */
