#include "hip/hip_runtime.h"
#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/cuda/impl/cuda_buffer_generic_obj_kernel.cuh"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

#if !defined( SIXTRL_NO_SYSTEM_INCLUDES )
    #include <hip/hip_runtime.h>
    #include <hip/hip_runtime.h>
#endif /* !defined( SIXTRL_NO_SYSTEM_INCLUDES ) */

#if !defined( SIXTRL_NO_INCLUDES )
    #include "sixtracklib/_impl/definitions.h"
    #include "sixtracklib/common/impl/managed_buffer_minimal.h"
    #include "sixtracklib/common/impl/managed_buffer_remap.h"
    #include "sixtracklib/testlib/generic_buffer_obj.h"
    #include "sixtracklib/cuda/impl/cuda_tools.h"
#endif /* !defined( SIXTRL_NO_INCLUDES ) */

extern __global__ void Remap_original_buffer_kernel_cuda(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT orig_begin,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT copy_begin,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag );

extern __global__ void NS(Copy_original_buffer_kernel_cuda)(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char const* SIXTRL_RESTRICT orig_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char*       SIXTRL_RESTRICT copy_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag );

extern __host__ int NS(Run_test_buffer_generic_obj_kernel_on_cuda)(
    dim3 const grid_dim, dim3 const block_dim,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char const* SIXTRL_RESTRICT orig_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT copy_buffer );


__global__ void NS(Remap_original_buffer_kernel_cuda)(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT orig_begin,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT copy_begin,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag )
{
    typedef NS(buffer_size_t) buf_size_t;

    buf_size_t const thread_id =
        NS(Cuda_get_1d_thread_id_in_kernel)();

    buf_size_t const total_num_threads =
        NS(Cuda_get_total_num_threads_in_kernel)();

    buf_size_t const tid_to_remap_orig_buffer = ( buf_size_t )0u;

    buf_size_t const tid_to_remap_copy_buffer =
        ( total_num_threads > ( buf_size_t )1u )
            ? ( tid_to_remap_orig_buffer + ( buf_size_t )1u )
            : ( tid_to_remap_orig_buffer );

    if( thread_id <= tid_to_remap_copy_buffer )
    {
        int32_t success_flag = ( int32_t )0;
        buf_size_t const slot_size = ( buf_size_t )8u;

        if( thread_id == tid_to_remap_orig_buffer )
        {
            if( ( success_flag == 0 ) && ( orig_begin != SIXTRL_NULLPTR ) )
            {
                if( NS(ManagedBuffer_needs_remapping)( orig_begin, slot_size ) )
                {
                    if( NS(ManagedBuffer_remap)( orig_begin, slot_size ) != 0 )
                    {
                        success_flag |= -1;
                    }
                }
            }
            else if( orig_begin != SIXTRL_NULLPTR )
            {
                success_flag |= -2;
            }
        }

        if( thread_id == tid_to_remap_copy_buffer )
        {
            if( ( success_flag == 0 ) && ( copy_begin != SIXTRL_NULLPTR ) )
            {
                if( NS(ManagedBuffer_needs_remapping)( copy_begin, slot_size ) )
                {
                    if( NS(ManagedBuffer_remap)( copy_begin, slot_size ) != 0 )
                    {
                        success_flag |= -4;
                    }
                }
            }
            else if( copy_begin != SIXTRL_NULLPTR )
            {
                success_flag |= -8;
            }
        }

        if( ( success_flag != 0 ) && ( ptr_success_flag != SIXTRL_NULLPTR ) )
        {
            #if defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 )
            atomicOr( ptr_success_flag, success_flag );
            #else /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 ) */
            *ptr_success_flag |= success_flag;
            #endif /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 ) */
        }
    }

    return;
}

__global__ void NS(Copy_original_buffer_kernel_cuda)(
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char const* SIXTRL_RESTRICT orig_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char*       SIXTRL_RESTRICT copy_buffer,
    SIXTRL_BUFFER_DATAPTR_DEC int32_t* SIXTRL_RESTRICT ptr_success_flag )
{
    typedef NS(buffer_size_t) buf_size_t;

    buf_size_t work_item_id =
        NS(Cuda_get_1d_thread_id_in_kernel)();

    buf_size_t const total_num_threads =
        NS(Cuda_get_total_num_threads_in_kernel)();

    buf_size_t const stride =
        NS(Cuda_get_1d_thread_stride_in_kernel)();

    buf_size_t const slot_size = ( buf_size_t )8u;

    int32_t success_flag = -1;

    if( ( !NS(ManagedBuffer_needs_remapping)( orig_buffer, slot_size ) ) &&
        ( !NS(ManagedBuffer_needs_remapping)( copy_buffer, slot_size ) ) )
    {
        typedef NS(Object)      object_t;
        typedef NS(GenericObj)  gen_obj_t;

        typedef SIXTRL_BUFFER_OBJ_ARGPTR_DEC object_t const*    in_index_ptr_t;
        typedef SIXTRL_BUFFER_OBJ_ARGPTR_DEC object_t*          out_index_ptr_t;

        typedef SIXTRL_BUFFER_OBJ_DATAPTR_DEC gen_obj_t const* in_obj_ptr_t;
        typedef SIXTRL_BUFFER_OBJ_DATAPTR_DEC gen_obj_t*       out_obj_ptr_t;

        buf_size_t const num_obj = NS(ManagedBuffer_get_num_objects)(
            orig_buffer, slot_size );

        success_flag = 0;

        while( ( work_item_id < num_obj ) && ( success_flag == 0 ) )
        {
            in_index_ptr_t ptr_in_info = ( in_index_ptr_t )( uintptr_t
                )NS(ManagedBuffer_get_const_objects_index_begin)(
                    orig_buffer, slot_size );

            out_index_ptr_t ptr_out_info = ( out_index_ptr_t )( uintptr_t
               )NS(ManagedBuffer_get_objects_index_begin)(
                   copy_buffer, slot_size );

            in_obj_ptr_t  in_obj  = SIXTRL_NULLPTR;
            out_obj_ptr_t out_obj = SIXTRL_NULLPTR;

            success_flag = ( ( ptr_in_info  != SIXTRL_NULLPTR ) &&
                             ( ptr_out_info != SIXTRL_NULLPTR ) ) ? 0 : -2;

            ptr_in_info   = ptr_in_info  + work_item_id;
            ptr_out_info  = ptr_out_info + work_item_id;
            work_item_id += stride;

            in_obj  = ( in_obj_ptr_t  )( uintptr_t
                )NS(Object_get_const_begin_ptr)( ptr_in_info );

            out_obj = ( out_obj_ptr_t )( uintptr_t
                )NS(Object_get_begin_ptr)( ptr_out_info );

            if( ( out_obj != SIXTRL_NULLPTR ) && ( in_obj != SIXTRL_NULLPTR ) &&
                ( out_obj != in_obj ) &&
                ( out_obj->type_id == in_obj->type_id ) &&
                ( out_obj->num_d   == in_obj->num_d   ) &&
                ( out_obj->num_e   == in_obj->num_e   ) &&
                ( out_obj->d != in_obj->d ) &&
                ( out_obj->d != SIXTRL_NULLPTR ) &&
                ( in_obj->d  != SIXTRL_NULLPTR ) &&
                ( out_obj->e != SIXTRL_NULLPTR ) &&
                ( in_obj->e  != SIXTRL_NULLPTR ) )
            {
                out_obj->a = in_obj->a;
                out_obj->b = in_obj->b;

                SIXTRACKLIB_COPY_VALUES( SIXTRL_REAL_T,
                     &out_obj->c[ 0 ], &in_obj->c[ 0 ], ( size_t )4u );

                SIXTRACKLIB_COPY_VALUES( SIXTRL_UINT8_T,
                     out_obj->d, in_obj->d, in_obj->num_d );

                SIXTRACKLIB_COPY_VALUES( SIXTRL_REAL_T,
                     out_obj->e, in_obj->e, in_obj->num_e );
            }
            else
            {
               success_flag |= -4;
            }
        }
    }

    if( ( success_flag != 0 ) && ( ptr_success_flag != SIXTRL_NULLPTR ) )
    {
        #if defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 )
        atomicOr( ptr_success_flag, success_flag );
        #else /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 ) */
        *ptr_success_flag |= success_flag;
        #endif /* defined( __CUDA_ARCH__ ) && ( __CUDA_ARCH__ > 120 ) */
    }

    return;
}

__host__ int NS(Run_test_buffer_generic_obj_kernel_on_cuda)(
    dim3 const grid_dim, dim3 const block_dim,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char const* SIXTRL_RESTRICT orig_buffer_begin,
    SIXTRL_BUFFER_DATAPTR_DEC unsigned char* SIXTRL_RESTRICT copy_buffer_begin )
{
    typedef NS(buffer_size_t) buf_size_t;

    int success = -16;

    buf_size_t const slot_size = ( buf_size_t )8u;

    size_t const orig_buffer_size = NS(ManagedBuffer_get_buffer_length)(
        orig_buffer_begin, slot_size );

    size_t const copy_buffer_size = NS(ManagedBuffer_get_buffer_length)(
        copy_buffer_begin, slot_size );

    if( ( orig_buffer_begin != SIXTRL_NULLPTR ) &&
        ( copy_buffer_begin != SIXTRL_NULLPTR ) &&
        ( orig_buffer_size > slot_size ) &&
        ( orig_buffer_size == copy_buffer_size ) )
    {
        int32_t success_flag = 0;

        unsigned char* cuda_orig_begin = nullptr;
        unsigned char* cuda_copy_begin = nullptr;
        int32_t* cuda_success_flag     = nullptr;

        SIXTRL_ASSERT( orig_buffer_size == copy_buffer_size );

        SIXTRL_ASSERT( NS(ManagedBuffer_get_num_objects)( orig_buffer_begin,
            slot_size ) == NS(ManagedBuffer_get_num_of_objects)(
                copy_buffer_begin, slot_size ) );

        SIXTRL_ASSERT( !NS(ManagedBuffer_needs_remapping)(
            orig_buffer_begin, slot_size ) );

        SIXTRL_ASSERT( !NS(ManagedBuffer_needs_remapping)(
            copy_buffer_begin, slot_size ) );

        if( hipSuccess == hipMalloc(
            ( void** )&cuda_orig_begin, orig_buffer_size ) )
        {
            success = 0;
        }

        if( ( success == 0 ) && ( hipSuccess != hipMalloc(
                ( void** )&cuda_copy_begin, copy_buffer_size ) ) )
        {
            success |= -32;
        }

        if( ( success == 0 ) && ( hipSuccess != hipMalloc(
                ( void** )&cuda_success_flag, sizeof( success_flag ) ) ) )
        {
            success |= -64;
        }

        SIXTRL_ASSERT( ( success != 0 ) ||
            ( cuda_orig_begin != SIXTRL_NULLPTR ) &&
            ( cuda_copy_begin != SIXTRL_NULLPTR ) &&
            ( cuda_success_flag != SIXTRL_NULLPTr ) );

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( cuda_orig_begin, orig_buffer_begin,
                orig_buffer_size, hipMemcpyHostToDevice ) ) )
        {
            success |= -128;
        }

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( cuda_copy_begin, copy_buffer_begin,
                copy_buffer_size, hipMemcpyHostToDevice ) ) )
        {
            success |= -256;
        }

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( cuda_success_flag, &success_flag,
                sizeof( success_flag ), hipMemcpyHostToDevice ) ) )
        {
            success |= -512;
        }


        if( success == 0 )
        {
            NS(Remap_original_buffer_kernel_cuda)<<< grid_dim, block_dim >>>(
                cuda_orig_begin, cuda_copy_begin, cuda_success_flag );

            if( hipSuccess != hipDeviceSynchronize() )
            {
                success |= -1024;
            }
        }

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( &success_flag, cuda_success_flag,
                sizeof( success_flag ), hipMemcpyDeviceToHost ) ) )
        {
            success |= -2048;
        }

        if( success == 0 )
        {
            success |= ( int )success_flag;
        }

        if( success == 0 )
        {
            NS(Copy_original_buffer_kernel_cuda)<<< grid_dim, block_dim >>>(
                cuda_orig_begin, cuda_copy_begin, cuda_success_flag );

            if( hipSuccess != hipDeviceSynchronize() )
            {
                success |= -4096;
            }
        }

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( &success_flag, cuda_success_flag,
                sizeof( success_flag ), hipMemcpyDeviceToHost ) ) )
        {
            success |= -8192;
        }

        if( success == 0 )
        {
            success |= ( int )success_flag;
        }

        if( ( success == 0 ) &&
            ( hipSuccess != hipMemcpy( copy_buffer_begin, cuda_copy_begin,
                copy_buffer_size, hipMemcpyDeviceToHost ) ) )
        {
            success |= -16384;
        }

        if( ( success == 0 ) &&
            ( 0 != NS(ManagedBuffer_remap)( copy_buffer_begin, slot_size ) ) )
        {
            success |= -32768;
        }

        if( ( ( cuda_orig_begin != SIXTRL_NULLPTR ) &&
              ( hipSuccess != hipFree( cuda_orig_begin ) ) ) ||
            ( ( cuda_copy_begin != SIXTRL_NULLPTR ) &&
              ( hipSuccess != hipFree( cuda_copy_begin ) ) ) ||
            ( ( cuda_success_flag != SIXTRL_NULLPTR ) &&
              ( hipSuccess != hipFree( cuda_success_flag ) ) ) )
        {
            success |= -65536;
        }
    }

    return success;
}

/* end: tests/sixtracklib/cuda/details/cuda_buffer_generic_obj_kernel.cu */
